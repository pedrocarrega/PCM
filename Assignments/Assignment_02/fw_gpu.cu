#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "include/time.h"

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} \

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

__global__ void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
    //TODO
    //__shared__ int cache[];
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
    
    
  struct timeval tv1, tv2, tv;
  float time_delta = 0;

  int *graph, *graph_gpu, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  //TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  //TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  //TIMER_START();
  
  HANDLE_ERROR((&graph_gpu, size));
  HANDLE_ERROR(hipMemcpy(graph_gpu, graph, size, hipMemcpyHostToDevice));

  int test = prop.maxThreadsPerBlock;

  floyd_warshall_gpu<<<size/test, test>>>(graph_gpu, GRAPH_SIZE, output_gpu);
  //TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  }

  return 0;
}
