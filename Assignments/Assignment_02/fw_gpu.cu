#include <assert.h>
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define GRAPH_SIZE 32

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} \

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

//__device__ int min(int x, int y) { return x < y ? x : y; }

__global__ void floyd_warshall_gpu(int *output, int graph_size, int const k) {
    
    //__shared__ int best;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= graph_size){
    //int idx = graph_size * blockIdx.y + col;
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    /*
    PLACE SHARED MEMORY
    __syncthreads();
    */
    if (D(col, k) + D(k, idx) < D(col, idx)) {
        D(col, idx) = D(col, k) + D(k, idx);
    }
    }
    

    //D(col, idx) = min(D(col, k), D(k, idx));
    

}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

void printGraph(int* output, int graph_size) {
    int i, j;
        for (i = 0; i < graph_size; i++) {
            
            for (j = 0; j < graph_size; j++) {
                int k = D(i, j);
                fprintf(stderr, "%d ", k);
            }
            fprintf(stderr, "\n");
        
    }
}

int main(int argc, char **argv) {
/*#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  
    
    
  struct timeval tv1, tv2, tv;
  float time_delta = 0;*/
    
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  int *graph, *graph_gpu, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  //TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  //TIMER_STOP();
  //fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  //TIMER_START();
  printGraph(output_cpu, GRAPH_SIZE);
  
  HANDLE_ERROR(hipMalloc(&graph_gpu, size));
  HANDLE_ERROR(hipMemcpy(graph_gpu, graph, size, hipMemcpyHostToDevice));

  dim3 dimGrid((GRAPH_SIZE + prop.maxThreadsPerBlock - 1) / prop.maxThreadsPerBlock, GRAPH_SIZE);

  for (int k = 0; k < GRAPH_SIZE; k++)
  {
      floyd_warshall_gpu<<<1, dim3(GRAPH_SIZE,GRAPH_SIZE)>>>(graph_gpu, GRAPH_SIZE, k);
      hipError_t err = hipDeviceSynchronize();
      if (err != hipSuccess) { printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); }
  }

  hipMemcpy(output_gpu, graph_gpu, size, hipMemcpyDeviceToHost);
  printGraph(output_gpu, GRAPH_SIZE);
  hipFree(graph_gpu);

  
  //TIMER_STOP();
  //fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  }

  return 0;
}
