#include "hip/hip_runtime.h"
#include <stdio.h>
#include "workshop.h"

#define N 1000

#define THREADS_PER_BLOCK 256
/* we launch at most 32 blocks in an attempt to keep the GPU busy
   this value is somewhat arbitrary and should be tweaked for performance */
#define BLOCKS MIN(32, (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

void print_vector(int *v)
{
    int c;
    for(c = 0; c < N; c++)
    {
        printf("%2d ", v[c]);
    }

    printf("\n");
}

void fill_vectors(int *a, int *b)
{
    int c;

    for(c = 0; c < N; c++)
    {
        a[c] = c + 1;
        b[c] = c + 1;
    }
}

__global__ void dot(int *a, int *b, int *temp, int *c)
{
    int outputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int i = outputIndex;
    int result = 0;

    /* multiplication step: compute partial sum */
    while(i < N)
    {
        result += a[i] * b[i];
        i += blockDim.x * gridDim.x;
    }

    temp[outputIndex] = result;

    /* wait for all threads to be done multiplying */
    __syncthreads();

    /* reduction step: sum all entries in the block and write to c */
    /* this requires that blockDim.x be a power of two! */
    i = blockDim.x / 2;
    while (i != 0)
    {
        /* only threads 0 through i are busy */
        if (threadIdx.x < i)
        {
            /* sum our output element with the one half a block away */
            temp[outputIndex] += temp[outputIndex + i];
        }

        /* wait for all threads within the block */
        __syncthreads();

        i /= 2;
    }

    /* thread 0 writes the results for this block */
    if (threadIdx.x == 0)
    {
        c[blockIdx.x] = temp[outputIndex];
    }
}

int main(int argc, char **argv)
{
    int host_a[N], host_b[N], host_c[BLOCKS];
    int *dev_a, *dev_b, *dev_tmp, *dev_c;
    int i;

    fill_vectors(host_a, host_b);

    HANDLE_ERROR( hipMalloc(&dev_a, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_b, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_tmp, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_c, sizeof(int) * BLOCKS) );

    hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);

    dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_tmp, dev_c);
    hipMemcpy(host_c, dev_c, sizeof(int) * BLOCKS, hipMemcpyDeviceToHost);

    /* finish the sum on the CPU */
    for(i = 1; i < BLOCKS; i++)
    {
        host_c[0] += host_c[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_tmp);
    hipFree(dev_c);

    printf("%d\n", host_c[0]);
    if (host_c[0] != N * (N + 1) * (2 * N + 1) / 6)
    {
        printf("FAIL!\n");
    }
}
