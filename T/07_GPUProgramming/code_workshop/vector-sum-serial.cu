#include "hip/hip_runtime.h"
#include <stdio.h>
#include "workshop.h"

#define N 1000

void print_vector(int *v)
{
    int c;
    for(c = 0; c < N; c++)
    {
        printf("%2d ", v[c]);
    }

    printf("\n");
}

void fill_vectors(int *a, int *b)
{
    int c;

    for(c = 0; c < N; c++)
    {
        a[c] = c;
        b[c] = N - c;
    }
}

__global__ void sum(int *a, int *b, int *c)
{
    int i;

    for(i = 0; i < N; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char **argv)
{
    int host_a[N], host_b[N], host_c[N];
    int *dev_a, *dev_b, *dev_c;
    int i;

    fill_vectors(host_a, host_b);

    HANDLE_ERROR( hipMalloc(&dev_a, sizeof(int) * N));
    HANDLE_ERROR( hipMalloc(&dev_b, sizeof(int) * N));
    HANDLE_ERROR( hipMalloc(&dev_c, sizeof(int) * N));

    hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);

    sum<<<1, 1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(host_c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    print_vector(host_c);

    for(i = 0; i < N; i++)
    {
        if (host_c[i] != N)
        {
            printf("FAIL!\n");
            break;
        }
    }
}

