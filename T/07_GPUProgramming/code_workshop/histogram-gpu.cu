#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "workshop.h"

#define N 10000000

unsigned char *fill_random_buffer(int size)
{
    unsigned char *ret;
    int c;

    ret = (unsigned char *) malloc(size);
    assert(ret);

    for(c = 0; c < size; c++)
    {
        ret[c] = rand();
    }

    return ret;
}

__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
    __shared__ unsigned int cache[256];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    cache[threadIdx.x] = 0;
    __syncthreads();

    while(i < N)
    {
        atomicAdd(&cache[data[i]], 1);
        i += blockDim.x * gridDim.x;
    }

    __syncthreads();
    atomicAdd(&histogram[threadIdx.x], cache[threadIdx.x]);
}

int main(int argc, char **argv)
{
    unsigned char *data = (unsigned char *) fill_random_buffer(N);
    unsigned int histogram[256];
    int c, sum;

    unsigned char *dev_data;
    unsigned int *dev_histogram;

    HANDLE_ERROR( hipMalloc(&dev_data, N) );
    HANDLE_ERROR( hipMalloc(&dev_histogram, sizeof(unsigned int) * 256) );

    hipMemcpy(dev_data, data, N, hipMemcpyHostToDevice);
    hipMemset(dev_histogram, 0, sizeof(unsigned int) * 256);

    compute_histogram<<<30, 256>>>(dev_data, dev_histogram);

    hipMemcpy(histogram, dev_histogram, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost);

    sum = 0;
    for(c = 0; c < 256; c++)
    {
        printf("%3d = %d\n", c, histogram[c]);
        sum += histogram[c];
    }

    if (sum != N)
    {
        printf("FAIL!\n");
    }

    free(data);
}
