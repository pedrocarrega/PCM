#include "hip/hip_runtime.h"
#include <stdio.h>
#include "workshop.h"

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main(int argc, char **argv)
{
    int c, *dev_c;
    HANDLE_ERROR( hipMalloc(&dev_c, sizeof(int)) );
    add<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR( hipMemcpy(&c, dev_c, sizeof(int),
                             hipMemcpyDeviceToHost) );
    printf("2 + 7 = %d\n", c);
}
