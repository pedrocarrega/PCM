#include "hip/hip_runtime.h"
#include <stdio.h>
#include "workshop.h"

#define N 1000

#define THREADS_PER_BLOCK 256
/* we launch at most 32 blocks in an attempt to keep the GPU busy
   this value is somewhat arbitrary and should be tweaked for performance */
#define BLOCKS MIN(32, (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

void print_vector(int *v)
{
    int c;
    for(c = 0; c < N; c++)
    {
        printf("%2d ", v[c]);
    }

    printf("\n");
}

void fill_vectors(int *a, int *b)
{
    int c;

    for(c = 0; c < N; c++)
    {
        a[c] = c + 1;
        b[c] = c + 1;
    }
}

__global__ void dot(int *a, int *b, int *c)
{
    /* shared memory cache for partial sum results */
    __shared__ int cache[THREADS_PER_BLOCK];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int result = 0;

    /* multiplication step: write a partial sum into the cache */
    while(i < N)
    {
        result += a[i] * b[i];
        i += blockDim.x * gridDim.x;
    }

    cache[threadIdx.x] = result;

    /* wait for all other threads in the same block */
    __syncthreads();

    /* reduction step: sum all entries in the cache */
    i = blockDim.x / 2;
    while (i != 0)
    {
        /* only threads 0 through i are busy */
        if (threadIdx.x < i)
        {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }

        /* wait for all threads within the block */
        __syncthreads();

        i /= 2;
    }

    /* thread 0 writes the result for this block */
    if (threadIdx.x == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}

int main(int argc, char **argv)
{
    int host_a[N], host_b[N], c[BLOCKS];
    int *dev_a, *dev_b, *dev_c;
    int i;

    fill_vectors(host_a, host_b);

    HANDLE_ERROR( hipMalloc(&dev_a, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_b, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_c, sizeof(int) * BLOCKS) );

    hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);

    dot<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, sizeof(int) * BLOCKS, hipMemcpyDeviceToHost);

    /* finish the sum on the CPU */
    for(i = 1; i < BLOCKS; i++)
    {
        c[0] += c[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf("%d\n", c[0]);
    if (c[0] != N * (N + 1) * (2 * N + 1) / 6)
    {
        printf("FAIL!\n");
    }
}
