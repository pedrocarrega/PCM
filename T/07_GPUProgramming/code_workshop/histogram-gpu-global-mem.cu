#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "workshop.h"

#define N 10000000

unsigned char *fill_random_buffer(int size)
{
    unsigned char *ret;
    int c;

    ret = (unsigned char *) malloc(size);
    assert(ret);

    for(c = 0; c < size; c++)
    {
        ret[c] = rand();
    }

    return ret;
}

__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < N)
    {
        atomicAdd(&histogram[data[i]], 1);
        i += blockDim.x * gridDim.x;
    }
}

int main(int argc, char **argv)
{
    unsigned char *data = (unsigned char *) fill_random_buffer(N);
    unsigned int histogram[256];
    int c, sum;

    unsigned char *dev_data;
    unsigned int *dev_histogram;

    HANDLE_ERROR( hipMalloc(&dev_data, N) );
    HANDLE_ERROR( hipMalloc(&dev_histogram, sizeof(unsigned int) * 256) );

    hipMemcpy(dev_data, data, N, hipMemcpyHostToDevice);
    hipMemset(dev_histogram, 0, sizeof(unsigned int) * 256);

    compute_histogram<<<30, 256>>>(dev_data, dev_histogram);

    hipMemcpy(histogram, dev_histogram, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost);

    sum = 0;
    for(c = 0; c < 256; c++)
    {
        printf("%3d = %d\n", c, histogram[c]);
        sum += histogram[c];
    }

    if (sum != N)
    {
        printf("FAIL!\n");
    }

    free(data);
}
