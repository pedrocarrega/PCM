#include "hip/hip_runtime.h"
#include <stdio.h>
#include "workshop.h"

#define N 1000

void print_vector(int *v)
{
    int c;
    for(c = 0; c < N; c++)
    {
        printf("%2d ", v[c]);
    }

    printf("\n");
}

void fill_vectors(int *a, int *b)
{
    int c;

    for(c = 0; c < N; c++)
    {
        a[c] = c + 1;
        b[c] = c + 1;
    }
}

__global__ void dot(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while(i < N)
    {
        c[i] = a[i] * b[i];
        i += blockDim.x * gridDim.x;
    }
}

int main(int argc, char **argv)
{
    int host_a[N], host_b[N], host_c[N];
    int *dev_a, *dev_b, *dev_c;
    int i;

    fill_vectors(host_a, host_b);

    HANDLE_ERROR( hipMalloc(&dev_a, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_b, sizeof(int) * N) );
    HANDLE_ERROR( hipMalloc(&dev_c, sizeof(int) * N) );

    hipMemcpy(dev_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(int) * N, hipMemcpyHostToDevice);

    dot<<<256, 256>>>(dev_a, dev_b, dev_c);

    hipMemcpy(host_c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for(i = 1; i < N; i++)
    {
        host_c[0] += host_c[i];
    }

    printf("%d\n", host_c[0]);
    if (host_c[0] != N * (N + 1) * (2 * N + 1) / 6)
    {
        printf("FAIL!\n");
    }
}
